/* -----------------------------------------------------------------
 * Programmer(s): Slaven Peles, and Cody J. Balos @ LLNL
 * -----------------------------------------------------------------
 * SUNDIALS Copyright Start
 * Copyright (c) 2002-2020, Lawrence Livermore National Security
 * and Southern Methodist University.
 * All rights reserved.
 *
 * See the top-level LICENSE and NOTICE files for details.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 * SUNDIALS Copyright End
 * -----------------------------------------------------------------
 * This is the testing routine to check the NVECTOR CUDA module
 * implementation.
 * -----------------------------------------------------------------*/

#include <stdio.h>
#include <stdlib.h>

#include <sundials/sundials_math.h>
#include <sundials/sundials_types.h>
#include <nvector/nvector_serial.h>
#include <nvector/nvector_cuda.h>
#include "test_nvector.h"

/* private custom allocator functions */
static void* sunalloc(size_t);
static void sunfree(void* ptr);

/* CUDA vector specific tests */
static int Test_N_VMake_Cuda(N_Vector X, sunindextype length, int myid);
static int Test_N_VMakeManaged_Cuda(N_Vector X, sunindextype length, int myid);
static int Test_N_VMakeWithManagedAllocator_Cuda(sunindextype length, int myid);

/* CUDA vector variants */
enum mem_type { UNMANAGED, MANAGED, CUSTOM };
enum pol_type { DEFAULT_POL, DEFAULT_POL_W_STREAM, GRID_STRIDE };

/* ----------------------------------------------------------------------
 * Main NVector Testing Routine
 * --------------------------------------------------------------------*/
int main(int argc, char *argv[])
{
  int          fails = 0;         /* counter for test failures */
  int          retval;            /* function return value     */
  sunindextype length;            /* vector length             */
  N_Vector     U, V, X, Y, Z;     /* test vectors              */
  int          print_timing;      /* turn timing on/off        */
  int          threadsPerBlock;   /* cuda block size           */
  hipStream_t stream;            /* cuda stream               */
  int          memtype, policy;


  /* check input and set vector length */
  if (argc < 4){ 
    printf("ERROR: THREE (3) Inputs required: vector length, CUDA threads per block (-1 for default), print timing \n");
    return(-1);
  }

  length = (sunindextype) atol(argv[1]);
  if (length <= 0) {
    printf("ERROR: length of vector must be a positive integer\n");
    return(-1);
  }

  threadsPerBlock = (int) atoi(argv[2]);
  if (threadsPerBlock != -1 && threadsPerBlock % 32) {
    printf("ERROR: CUDA threads per block must be -1 to use the default or a multiple of 32\n");
    return(-1);
  }

  print_timing = atoi(argv[3]);
  SetTiming(print_timing, 0);

  /* test with all policy variants */
  for (policy=DEFAULT_POL; policy<=GRID_STRIDE; ++policy) {
    int actualThreadsPerBlock = (threadsPerBlock == -1) ? 256 : threadsPerBlock;
    SUNCudaExecPolicy* stream_exec_policy = NULL;
    SUNCudaExecPolicy* reduce_exec_policy = NULL;
    hipStreamCreate(&stream);

    if (policy == DEFAULT_POL_W_STREAM) {
      stream_exec_policy = new SUNCudaThreadDirectExecPolicy(actualThreadsPerBlock, stream);
      reduce_exec_policy = new SUNCudaBlockReduceExecPolicy(actualThreadsPerBlock, 0, stream);
    } else if (policy == GRID_STRIDE) {
      stream_exec_policy = new SUNCudaGridStrideExecPolicy(actualThreadsPerBlock, 1);
      reduce_exec_policy = new SUNCudaBlockReduceExecPolicy(actualThreadsPerBlock, 1);
    }

    /* test with all memory variants */
    for (memtype=UNMANAGED; memtype<=CUSTOM; ++memtype) {
      printf("=====> Beginning setup\n\n");

      if (memtype==UNMANAGED) {
        printf("Testing CUDA N_Vector, policy %d\n", policy);
      } else if (memtype==MANAGED) {
        printf("Testing CUDA N_Vector with managed memory, policy %d\n", policy);
      } else if (memtype==CUSTOM) {
        printf("Testing CUDA N_Vector with custom allocator, policy %d\n", policy);
      }
      printf("Vector length: %ld \n", (long int) length);

      /* Create new vectors */
      if (memtype == UNMANAGED)    X = N_VNew_Cuda(length);
      else if (memtype == MANAGED) X = N_VNewManaged_Cuda(length);
      else if (memtype == CUSTOM)  X = N_VMakeWithManagedAllocator_Cuda(length, sunalloc, sunfree);
      if (X == NULL) {
        delete stream_exec_policy;
        delete reduce_exec_policy;
        printf("FAIL: Unable to create a new vector \n\n");
        return(1);
      }

      if (stream_exec_policy != NULL && reduce_exec_policy != NULL) {
        if (N_VSetKernelExecPolicy_Cuda(X, stream_exec_policy, reduce_exec_policy)) {
          N_VDestroy(X);
          delete stream_exec_policy;
          delete reduce_exec_policy;
          printf("FAIL: Unable to set kernel execution policy \n\n");
          return(1);
        }
        printf("Using non-default kernel execution policy\n");
        printf("Threads per block: %d\n\n", actualThreadsPerBlock);
      }

      /* Fill vector with uniform random data in [-1,1] */
      realtype* xdata = N_VGetHostArrayPointer_Cuda(X);
      for (sunindextype j=0; j<length; j++)
        xdata[j] = ((realtype) rand() / (realtype) RAND_MAX)*2-1;
      N_VCopyToDevice_Cuda(X);

      /* Clone additional vectors for testing */
      Y = N_VClone(X);
      if (Y == NULL) {
        N_VDestroy(X);
        printf("FAIL: Unable to create a new vector \n\n");
        delete stream_exec_policy;
        delete reduce_exec_policy;
        return(1);
      }

      Z = N_VClone(X);
      if (Z == NULL) {
        N_VDestroy(X);
        N_VDestroy(Y);
        delete stream_exec_policy;
        delete reduce_exec_policy;
        printf("FAIL: Unable to create a new vector \n\n");
        return(1);
      }

      /* Fill vectors with uniform random data in [-1,1] */
      realtype* ydata = N_VGetHostArrayPointer_Cuda(Y);
      realtype* zdata = N_VGetHostArrayPointer_Cuda(Z);
      for (sunindextype j=0; j<length; j++) {
        ydata[j] = ((realtype) rand() / (realtype) RAND_MAX)*2-1;
        zdata[j] = ((realtype) rand() / (realtype) RAND_MAX)*2-1;
      }
      N_VCopyToDevice_Cuda(Y);
      N_VCopyToDevice_Cuda(Z);

      printf("=====> Setup complete\n");
      printf("=====> Beginning tests\n\n");

      /* Standard vector operation tests */
      printf("\nTesting standard vector operations:\n\n");

      /* Check vector ID */
      fails += Test_N_VGetVectorID(X, SUNDIALS_NVEC_CUDA, 0);

      /* Check vector length */
      fails += Test_N_VGetLength(X, 0);

      /* Check vector communicator */
      fails += Test_N_VGetCommunicator(X, NULL, 0);

      /* Test clone functions */
      fails += Test_N_VCloneEmpty(X, 0);
      fails += Test_N_VClone(X, length, 0);
      fails += Test_N_VCloneEmptyVectorArray(5, X, 0);
      fails += Test_N_VCloneVectorArray(5, X, length, 0);

      /* Test vector math kernels */
      fails += Test_N_VConst(X, length, 0);
      fails += Test_N_VLinearSum(X, Y, Z, length, 0);
      fails += Test_N_VProd(X, Y, Z, length, 0);
      fails += Test_N_VDiv(X, Y, Z, length, 0);
      fails += Test_N_VScale(X, Z, length, 0);
      fails += Test_N_VAbs(X, Z, length, 0);
      fails += Test_N_VInv(X, Z, length, 0);
      fails += Test_N_VAddConst(X, Z, length, 0);
      fails += Test_N_VDotProd(X, Y, length, 0);
      fails += Test_N_VMaxNorm(X, length, 0);
      fails += Test_N_VWrmsNorm(X, Y, length, 0);
      fails += Test_N_VWrmsNormMask(X, Y, Z, length, 0);
      fails += Test_N_VMin(X, length, 0);
      fails += Test_N_VWL2Norm(X, Y, length, 0);
      fails += Test_N_VL1Norm(X, length, 0);
      if (length >= 3) fails += Test_N_VCompare(X, Z, length, 0);
      fails += Test_N_VInvTest(X, Z, length, 0);
      if (length >= 7) fails += Test_N_VConstrMask(X, Y, Z, length, 0);
      fails += Test_N_VMinQuotient(X, Y, length, 0);

      /* Fused and vector array operations tests (disabled) */
      printf("\nTesting fused and vector array operations (disabled):\n\n");

      /* create vector and disable all fused and vector array operations */
      if (memtype == UNMANAGED)    U = N_VNew_Cuda(length);
      else if (memtype == MANAGED) U = N_VNewManaged_Cuda(length);
      else                   U = N_VMakeWithManagedAllocator_Cuda(length, sunalloc, sunfree);
      if (U == NULL) {
        N_VDestroy(X);
        N_VDestroy(Y);
        delete stream_exec_policy;
        delete reduce_exec_policy;
        printf("FAIL: Unable to create a new vector \n\n");
        return(1);
      }
      retval = N_VEnableFusedOps_Cuda(U, SUNFALSE);
      if (retval != 0) {
        N_VDestroy(X);
        N_VDestroy(Y);
        N_VDestroy(Z);
        N_VDestroy(U);
        delete stream_exec_policy;
        delete reduce_exec_policy;
        printf("FAIL: Unable to create a new vector \n\n");
        return(1);
      }

      /* fused operations */
      fails += Test_N_VLinearCombination(U, length, 0);
      fails += Test_N_VScaleAddMulti(U, length, 0);
      fails += Test_N_VDotProdMulti(U, length, 0);

      /* vector array operations */
      fails += Test_N_VLinearSumVectorArray(U, length, 0);
      fails += Test_N_VScaleVectorArray(U, length, 0);
      fails += Test_N_VConstVectorArray(U, length, 0);
      fails += Test_N_VWrmsNormVectorArray(U, length, 0);
      fails += Test_N_VWrmsNormMaskVectorArray(U, length, 0);
      fails += Test_N_VScaleAddMultiVectorArray(U, length, 0);
      fails += Test_N_VLinearCombinationVectorArray(U, length, 0);

      /* Fused and vector array operations tests (enabled) */
      printf("\nTesting fused and vector array operations (enabled):\n\n");

      /* create vector and enable all fused and vector array operations */
      if (memtype == UNMANAGED)    V = N_VNew_Cuda(length);
      else if (memtype == MANAGED) V = N_VNewManaged_Cuda(length);
      else                         V = N_VMakeWithManagedAllocator_Cuda(length, sunalloc, sunfree);
      retval = N_VEnableFusedOps_Cuda(V, SUNTRUE);
      if (V == NULL) {
        N_VDestroy(X);
        N_VDestroy(Y);
        N_VDestroy(Z);
        N_VDestroy(U);
        printf("FAIL: Unable to create a new vector \n\n");
        delete stream_exec_policy;
        delete reduce_exec_policy;
        return(1);
      }
      if (retval != 0) {
        N_VDestroy(X);
        N_VDestroy(Y);
        N_VDestroy(Z);
        N_VDestroy(U);
        N_VDestroy(V);
        delete stream_exec_policy;
        delete reduce_exec_policy;
        printf("FAIL: Unable to create a new vector \n\n");
        return(1);
      }

      /* fused operations */
      fails += Test_N_VLinearCombination(V, length, 0);
      fails += Test_N_VScaleAddMulti(V, length, 0);
      fails += Test_N_VDotProdMulti(V, length, 0);

      /* vector array operations */
      fails += Test_N_VLinearSumVectorArray(V, length, 0);
      fails += Test_N_VScaleVectorArray(V, length, 0);
      fails += Test_N_VConstVectorArray(V, length, 0);
      fails += Test_N_VWrmsNormVectorArray(V, length, 0);
      fails += Test_N_VWrmsNormMaskVectorArray(V, length, 0);
      fails += Test_N_VScaleAddMultiVectorArray(V, length, 0);
      fails += Test_N_VLinearCombinationVectorArray(V, length, 0);

      /* local reduction operations */
      printf("\nTesting local reduction operations:\n\n");

      fails += Test_N_VDotProdLocal(X, Y, length, 0);
      fails += Test_N_VMaxNormLocal(X, length, 0);
      fails += Test_N_VMinLocal(X, length, 0);
      fails += Test_N_VL1NormLocal(X, length, 0);
      fails += Test_N_VWSqrSumLocal(X, Y, length, 0);
      fails += Test_N_VWSqrSumMaskLocal(X, Y, Z, length, 0);
      fails += Test_N_VInvTestLocal(X, Z, length, 0);
      if (length >= 7) fails += Test_N_VConstrMaskLocal(X, Y, Z, length, 0);
      fails += Test_N_VMinQuotientLocal(X, Y, length, 0);

      /* CUDA specific tests */
      printf("\nTesting cuda vector specific operations:\n\n");
      if (memtype==UNMANAGED) {
        fails += Test_N_VMake_Cuda(X, length, 0);
      } else if (memtype==MANAGED) {
        fails += Test_N_VMakeManaged_Cuda(X, length, 0);
      } else if (memtype==CUSTOM) {
        fails += Test_N_VMakeWithManagedAllocator_Cuda(length, 0);
      }

      printf("\n=====> Beginning teardown\n");

      /* Free vectors */
      N_VDestroy(X);
      N_VDestroy(Y);
      N_VDestroy(Z);
      N_VDestroy(U);
      N_VDestroy(V);

      /* Synchronize */
      hipDeviceSynchronize();

      printf("=====> Teardown complete\n\n");
    }

    /* Print result */
    if (fails) {
      printf("\n\nFAIL: NVector module failed %i tests \n\n", fails);
    } else {
      printf("\n\nSUCCESS: NVector module passed all tests \n\n");
    }

    hipStreamDestroy(stream);
    delete stream_exec_policy;
    delete reduce_exec_policy;
  }
  
  hipDeviceSynchronize();
  hipDeviceReset();
  return(fails);
}


/* ----------------------------------------------------------------------
 * CUDA specific tests
 * --------------------------------------------------------------------*/

/* --------------------------------------------------------------------
 * Test for the CUDA N_Vector N_VMake_Cuda function. Requires N_VConst
 * to check data.
 */
int Test_N_VMake_Cuda(N_Vector X, sunindextype length, int myid)
{
  int failure = 0;
  realtype *h_data, *d_data;
  N_Vector Y;

  N_VConst(NEG_HALF, X);
  N_VCopyFromDevice_Cuda(X);

  h_data = N_VGetHostArrayPointer_Cuda(X);
  d_data = N_VGetDeviceArrayPointer_Cuda(X);

  /* Case 1: h_data and d_data are not null */
  Y = N_VMake_Cuda(length, h_data, d_data);
  if (Y == NULL) {
    printf(">>> FAILED test -- N_VMake_Cuda, Proc %d \n", myid);
    printf("    Vector is NULL \n \n");
    return(1);
  }

  if (N_VGetHostArrayPointer_Cuda(Y) == NULL) {
    printf(">>> FAILED test -- N_VMake_Cuda, Proc %d \n", myid);
    printf("    Vector host data == NULL \n \n");
    N_VDestroy(Y);
    return(1);
  }

  if (N_VGetDeviceArrayPointer_Cuda(Y) == NULL) {
    printf(">>> FAILED test -- N_VMake_Cuda, Proc %d \n", myid);
    printf("    Vector device data -= NULL \n \n");
    N_VDestroy(Y);
    return(1);
  }

  failure += check_ans(NEG_HALF, Y, length);

  if (failure) {
    printf(">>> FAILED test -- N_VMake_Cuda Case 1, Proc %d \n", myid);
    printf("    Failed N_VConst check \n \n");
    N_VDestroy(Y);
    return(1);
  }

  if (myid == 0) {
    printf("PASSED test -- N_VMake_Cuda Case 1 \n");
  }

  N_VDestroy(Y);

  /* Case 2: data is null */
  Y = N_VMake_Cuda(length, NULL, NULL);
  if (Y != NULL) {
    printf(">>> FAILED test -- N_VMake_Cuda Case 2, Proc %d \n", myid);
    printf("    Vector is not NULL \n \n");
    return(1);
  }

  if (myid == 0) {
    printf("PASSED test -- N_VMake_Cuda Case 2 \n");
  }

  N_VDestroy(Y);

  return(failure);
}

/* --------------------------------------------------------------------
 * Test for the CUDA N_Vector N_VMakeManaged_Cuda function. Requires
 * N_VConst to check data. X must be using managed memory.
 */
int Test_N_VMakeManaged_Cuda(N_Vector X, sunindextype length, int myid)
{
  int failure = 0;
  realtype *vdata;
  N_Vector Y;

  if(!N_VIsManagedMemory_Cuda(X)) {
    printf(">>> FAILED test -- N_VIsManagedMemory_Cuda, Proc %d \n", myid);
    return(1);
  }

  N_VConst(NEG_HALF, X);
  vdata = N_VGetHostArrayPointer_Cuda(X);

  /* Case 1: data is not null */
  Y = N_VMakeManaged_Cuda(length, vdata);
  if (Y == NULL) {
    printf(">>> FAILED test -- N_VMakeManaged_Cuda, Proc %d \n", myid);
    printf("    Vector is NULL \n \n");
    return(1);
  }

  failure += check_ans(NEG_HALF, Y, length);
  if (failure) {
    printf(">>> FAILED test -- N_VMakeManaged_Cuda Case 1, Proc %d \n", myid);
    printf("    Failed N_VConst check \n \n");
    N_VDestroy(Y);
    return(1);
  }

  if (myid == 0) {
    printf("PASSED test -- N_VMakeManaged_Cuda Case 1\n");
  }

  N_VDestroy(Y);

  /* Case 2: data is null */
  Y = N_VMakeManaged_Cuda(length, NULL);
  if (Y != NULL) {
    printf(">>> FAILED test -- N_VMakeManaged_Cuda Case 2, Proc %d \n", myid);
    printf("    Vector is not NULL \n \n");
    return(1);
  }

  if (myid == 0) {
    printf("PASSED test -- N_VMakeManaged_Cuda Case 2 \n");
  }

  N_VDestroy(Y);

  return(failure);
}

/* --------------------------------------------------------------------
 * Test for the CUDA N_Vector N_VMakeWithManagedAllocator_Cuda function.
 * Requires N_VConst to check data. X must be using managed memory.
 */
int Test_N_VMakeWithManagedAllocator_Cuda(sunindextype length, int myid)
{
  int failure = 0;
  N_Vector Y;

  Y = N_VMakeWithManagedAllocator_Cuda(length, sunalloc, sunfree);
  if (Y == NULL) {
    printf(">>> FAILED test -- N_VMakeWithManagedAllocator_Cuda, Proc %d \n", myid);
    printf("    Vector is NULL \n \n");
    return(1);
  }

  N_VConst(NEG_HALF, Y);

  if(!N_VIsManagedMemory_Cuda(Y)) {
    printf(">>> FAILED test -- N_VMakeWithManagedAllocator_Cuda, Proc %d \n", myid);
    N_VDestroy(Y);
    return(1);
  }
  
  failure += check_ans(NEG_HALF, Y, length);
  if (failure) {
    printf(">>> FAILED test -- N_VMakeWithManagedAllocator_Cuda, Proc %d \n", myid);
    printf("    Failed N_VConst check \n \n");
    N_VDestroy(Y);
    return(1);
  }

  if (myid == 0) {
    printf("PASSED test -- N_VMakeWithManagedAllocator_Cuda\n");
  }
 
  N_VDestroy(Y);
 
  return(failure);
}

/* ----------------------------------------------------------------------
 * Implementation specific utility functions for vector tests
 * --------------------------------------------------------------------*/
int check_ans(realtype ans, N_Vector X, sunindextype length)
{
  int          failure = 0;
  sunindextype i;
  realtype     *Xdata;

  N_VCopyFromDevice_Cuda(X);
  Xdata = N_VGetHostArrayPointer_Cuda(X);

  /* check vector data */
  for (i = 0; i < length; i++) {
    if (failure += FNEQ(Xdata[i], ans)) {
      printf("check_ans fail: Xdata[%d] = %f, expected Xdata[%d] = %f\n", i, Xdata[i], i, ans);
    }
  }

  return (failure > ZERO) ? (1) : (0);
}

booleantype has_data(N_Vector X)
{
  /* check if vector data is non-null */
  if ((N_VGetHostArrayPointer_Cuda(X) == NULL) &&
      (N_VGetDeviceArrayPointer_Cuda(X) == NULL))
    return SUNFALSE;
  return SUNTRUE;
}

void set_element(N_Vector X, sunindextype i, realtype val)
{
  /* set i-th element of data array */
  set_element_range(X, i, i, val);
}

void set_element_range(N_Vector X, sunindextype is, sunindextype ie,
                       realtype val)
{
  sunindextype i;
  realtype*    xd;

  /* set elements [is,ie] of the data array */
  N_VCopyFromDevice_Cuda(X);
  xd = N_VGetHostArrayPointer_Cuda(X);
  for(i = is; i <= ie; i++) xd[i] = val;
  N_VCopyToDevice_Cuda(X);
}

realtype get_element(N_Vector X, sunindextype i)
{
  /* get i-th element of data array */
  N_VCopyFromDevice_Cuda(X);
  return (N_VGetHostArrayPointer_Cuda(X))[i];
}

double max_time(N_Vector X, double time)
{
  /* not running in parallel, just return input time */
  return(time);
}

void sync_device()
{
  /* sync with GPU */
  hipDeviceSynchronize();
  return;
}

void* sunalloc(size_t mem_size)
{
  void* ptr;
  hipError_t err;
  err = hipMallocManaged(&ptr, mem_size);
  if (err != hipSuccess) {
    printf("Error in sunalloc\n");
    ptr = NULL;
  }
  return ptr;
}

void sunfree(void* ptr)
{
  hipFree(ptr);
}
