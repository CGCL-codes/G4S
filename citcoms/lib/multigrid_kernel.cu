#include "hip/hip_runtime.h"
/* -*- C -*- */
/* vim:set ft=c: */

#if __CUDA_ARCH__ < 130
/* for double-precision floating-point */
#error This code requires compute capability 1.3 or higher; try giving "-arch sm_13".
#endif


#include "global_defs.h"
#include "element_definitions.h"
#include <assert.h>
#include <stdio.h>


enum {
    CAPS_PER_PROC = 1,
    M = 1, /* cap # */
    NSD = 3, /* Spatial extent: 3d */
    MAX_EQN = NSD*14,
};


struct Some_variables {
    int num_zero_resid;
    int *zero_resid;
    
    struct /*MESH_DATA*/ {
        int NEQ;
        int NNO;
    } lmesh;
    
    struct ID *ID;
    
    higher_precision *Eqn_k[NSD+1];
    int *Node_map;
    
    double *BI;
    
    double *temp;
    unsigned int *NODE;
    
    int2 *term;
};


/*------------------------------------------------------------------------*/

static void construct_E(
    struct Some_variables **d_E,
    struct Some_variables *s_E, /* host's shadow copy of d_E */
    struct Some_variables *E
    )
{
    /* construct a copy of 'E' in device memory */
    
    int neq = E->lmesh.NEQ;
    int nno = E->lmesh.NNO;
    
    s_E->lmesh.NEQ = E->lmesh.NEQ;
    s_E->lmesh.NNO = E->lmesh.NNO;
    
    /* ID -- cf. allocate_common_vars()*/
    hipMalloc((void **)&s_E->ID, (nno+1)*sizeof(struct ID));
    hipMemcpy(s_E->ID, E->ID, (nno+1)*sizeof(struct ID), hipMemcpyHostToDevice);
    
    /* Eqn_k, Node_map -- cf. construct_node_maps() */
    size_t matrix = MAX_EQN * nno;
    s_E->Eqn_k[0] = 0;
    hipMalloc((void **)&s_E->Eqn_k[1], 3*matrix*sizeof(higher_precision));
    s_E->Eqn_k[2] = s_E->Eqn_k[1] + matrix;
    s_E->Eqn_k[3] = s_E->Eqn_k[2] + matrix;
    hipMemcpy(s_E->Eqn_k[1], E->Eqn_k[1], matrix*sizeof(higher_precision), hipMemcpyHostToDevice);
    hipMemcpy(s_E->Eqn_k[2], E->Eqn_k[2], matrix*sizeof(higher_precision), hipMemcpyHostToDevice);
    hipMemcpy(s_E->Eqn_k[3], E->Eqn_k[3], matrix*sizeof(higher_precision), hipMemcpyHostToDevice);
    hipMalloc((void **)&s_E->Node_map, matrix*sizeof(int));
    hipMemcpy(s_E->Node_map, E->Node_map, matrix*sizeof(int), hipMemcpyHostToDevice);
    
    /* BI -- cf. allocate_velocity_vars() */
    hipMalloc((void **)&s_E->BI, neq*sizeof(double));
    hipMemcpy(s_E->BI, E->BI, neq*sizeof(double), hipMemcpyHostToDevice);
    
    /* temp -- cf. allocate_velocity_vars() */
    hipMalloc((void **)&s_E->temp, (neq+1)*sizeof(double));
    hipMemcpy(s_E->temp, E->temp, (neq+1)*sizeof(double), hipMemcpyHostToDevice);
    
    /* NODE -- cf. allocate_common_vars() */
    hipMalloc((void **)&s_E->NODE, (nno+1)*sizeof(unsigned int));
    hipMemcpy(s_E->NODE, E->NODE, (nno+1)*sizeof(unsigned int), hipMemcpyHostToDevice);
    
    /* term */
    hipMalloc((void **)&s_E->term, (neq+1) * MAX_EQN * sizeof(int2));
    hipMemcpy(s_E->term, E->term, (neq+1) * MAX_EQN * sizeof(int2), hipMemcpyHostToDevice);
    
    /* E */
    hipMalloc((void**)d_E, sizeof(Some_variables));
    hipMemcpy(*d_E, s_E, sizeof(Some_variables), hipMemcpyHostToDevice);
    
    return;
}

static void destroy_E(
    struct Some_variables *d_E,
    struct Some_variables *s_E
    )
{
    hipFree(s_E->ID);
    hipFree(s_E->Eqn_k[1]);
    hipFree(s_E->Node_map);
    hipFree(s_E->BI);
    hipFree(s_E->temp);
    hipFree(s_E->NODE);
    hipFree(s_E->term);
    hipFree(d_E);
}


/*------------------------------------------------------------------------*/
/* from Element_calculations.c */

__global__ void n_assemble_del2_u(
    struct Some_variables *E,
    double *u, double *Au,
    int strip_bcs
    )
{
    int n = blockIdx.x + 1; /* 1 <= n <= E->lmesh.NNO */
    int doff = blockIdx.y + 1; /* 1 <= doff <= NSD */ 
    unsigned int tid = threadIdx.x; /* 0 <= tid < MAX_EQN */
    
    /* Each block writes one element of Au in global memory: Au[eqn]. */
    int eqn = E->ID[n].doff[doff]; /* XXX: Compute this value? */
    
    if (n == 1 && doff == 1 && tid == 0) {
        Au[E->lmesh.NEQ] = 0.0;
    }
    
    if (strip_bcs) {
        /* See get_bcs_id_for_residual(). */
        unsigned int flags = E->NODE[n];
        unsigned int vb;
        switch (doff) {
        case 1: vb = VBX; break; /* 0x2 */
        case 2: vb = VBY; break; /* 0x8 */
        case 3: vb = VBZ; break; /* 0x4 */
        }
        if (flags & vb) {
            /* no-op: Au[eqn] is zero */
            if (tid == 0) {
                Au[eqn] = 0.0;
            }
            /* XXX: Hundreds of blocks exit here (E->num_zero_resid).
               Does it matter? */
            return;
        }
    }
    
    /* The partial sum computed by this thread. */
    double acc;
    
    /* Part I: The terms here are easily derived from the block and
       thread indices. */
    {
        int e = n; /* 1 <= e <= E->lmesh.NNO */
        int i = (int)tid; /* 0 <= i < MAX_EQN */
        
        if (i < 3) {
            acc = 0.0;
        } else {
            int *C = E->Node_map + (e-1)*MAX_EQN;
            higher_precision *B = E->Eqn_k[doff]+(e-1)*MAX_EQN;
            double UU = u[C[i]];
            acc = B[i]*UU;
        }
    }
    
    /* Part II: These terms are more complicated. */
    {
        int2 *term = E->term + eqn*MAX_EQN;
        int2 pair = term[tid];
        int e = pair.x; /* 1 <= e <= E->lmesh.NNO */
        int i = pair.y; /* 0 <= i < MAX_EQN */
        
        if (i != -1) {
            /* XXX: Compute these values? */
            int eqn1 = E->ID[e].doff[1];
            int eqn2 = E->ID[e].doff[2];
            int eqn3 = E->ID[e].doff[3];
            
            double U1 = u[eqn1];
            double U2 = u[eqn2];
            double U3 = u[eqn3];
            
            higher_precision *B1, *B2, *B3;
            B1 = E->Eqn_k[1]+(e-1)*MAX_EQN;
            B2 = E->Eqn_k[2]+(e-1)*MAX_EQN;
            B3 = E->Eqn_k[3]+(e-1)*MAX_EQN;
            
            acc += B1[i]*U1 +
                   B2[i]*U2 +
                   B3[i]*U3;
        } else {
            /* XXX: A considerable number of threads idle here. */
        }
    }
    
    /* Reduce the partial sums for this block.
       Based on reduce2() in the CUDA SDK. */
    __shared__ double sum[MAX_EQN];
    sum[tid] = acc;
    __syncthreads();
    if (tid >= 32) {
        sum[tid-32] += sum[tid];
    }
    __syncthreads();
    for (unsigned int s = 16; s > 0; s >>= 1) {
        if (tid < s) {
            sum[tid] += sum[tid + s];
        }
        /* XXX: not always necessary */
        __syncthreads();
    }
    
    /* Each block writes one element of Au in global memory. */
    if (tid == 0) {
        Au[eqn] = sum[0];
    }
    
    return;
}


/*------------------------------------------------------------------------*/
/* These are based on the function from General_matrix_functions.c. */

__global__ void gauss_seidel_0(
    int neq,
    double *d0,
    double *Ad
    )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < neq) {
        d0[i] = Ad[i] = 0.0;
    }
}

__global__ void gauss_seidel_1(
    struct Some_variables *E,
    int neq, int nno,
    double *F, double *Ad
    )
{
    const double zeroo = 0.0;
    
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    if (i <= nno) {
        /* 1 <= i <= E->lmesh.NNO */
        int doff = blockIdx.y + 1; /* 1 <= doff <= NSD */ 
        int eqn = E->ID[i].doff[doff];
        
        if (E->NODE[i] & OFFSIDE) {
            E->temp[eqn] = (F[eqn] - Ad[eqn])*E->BI[eqn];
        } else {
            E->temp[eqn] = zeroo;
        }
        
        if (i == 1 && doff == 1) {
            E->temp[neq] = zeroo;
            Ad[neq] = zeroo;
        }
    }
}

__global__ void gauss_seidel_2a(
    struct Some_variables *E,
    double *F, double *Ad
    )
{
    const int nno = E->lmesh.NNO;
    
    int j = threadIdx.x + 3; /* 3 <= j < MAX_EQN */
    int doff = threadIdx.y + 1; /* 1 <= doff <= NSD */ 
    
    for (int i = 1; i <= nno; i++) {
        int eqn = E->ID[i].doff[doff];
        int *C = E->Node_map + (i-1)*MAX_EQN;
        higher_precision *B = E->Eqn_k[doff] + (i-1)*MAX_EQN;
        
        /* Ad on boundaries differs after the following operation, but
           no communications are needed yet, because boundary Ad will
           not be used for the G-S iterations for interior nodes */
        
        __shared__ double UU[MAX_EQN-3];
        if (threadIdx.y == 0) {
            UU[threadIdx.x] = E->temp[C[j]];
        }
        __syncthreads();
        __shared__ double Ad_eqn[NSD];
        if (threadIdx.x == 0) {
            Ad_eqn[threadIdx.y] = Ad[eqn];
        }
        __shared__ double sum[MAX_EQN-3][3];
        /*for (int j = 3; j < MAX_EQN; j++)*/ {
            sum[threadIdx.x][threadIdx.y] = B[j]*UU[threadIdx.x];
        }
        __syncthreads();
        /* Reduce. */
        if (threadIdx.x >= 32) {
            sum[threadIdx.x-32][threadIdx.y] += sum[threadIdx.x][threadIdx.y];
        }
        __syncthreads();
        for (unsigned int s = 16; s > 0; s >>= 1) {
            if (threadIdx.x < s) {
                sum[threadIdx.x][threadIdx.y] += sum[threadIdx.x + s][threadIdx.y];
            }
            /* XXX: not always necessary */
            __syncthreads();
        }
        
        if (threadIdx.x == 0) {
            Ad_eqn[threadIdx.y] += sum[0][threadIdx.y];
            Ad[eqn] = Ad_eqn[threadIdx.y];
            if (!(E->NODE[i] & OFFSIDE)) {
                E->temp[eqn] = (F[eqn] - Ad_eqn[threadIdx.y])*E->BI[eqn];
            }
        }
        __syncthreads();
    }
}


__global__ void gauss_seidel_2b(
    struct Some_variables *E,
    double *F, double *Ad
    )
{
    const int nno = E->lmesh.NNO;
    unsigned int tid = threadIdx.x;
    
    for (int i = 1; i <= nno; i++) {
        
        int eqn1 = E->ID[i].doff[1];
        int eqn2 = eqn1 + 1;
        int eqn3 = eqn2 + 1;
        
        int *C = E->Node_map + (i-1)*MAX_EQN;
        
        higher_precision *B1,*B2,*B3;
        B1 = E->Eqn_k[1] + (i-1)*MAX_EQN;
        B2 = E->Eqn_k[2] + (i-1)*MAX_EQN;
        B3 = E->Eqn_k[3] + (i-1)*MAX_EQN;
        
        /* Ad on boundaries differs after the following operation, but
           no communications are needed yet, because boundary Ad will
           not be used for the G-S iterations for interior nodes */
        
        double Ad_eqn1, Ad_eqn2, Ad_eqn3;
        if (tid == 0) {
            Ad_eqn1 = Ad[eqn1];
            Ad_eqn2 = Ad[eqn2];
            Ad_eqn3 = Ad[eqn3];
        }
        
        __shared__ double sum1[MAX_EQN-3];
        __shared__ double sum2[MAX_EQN-3];
        __shared__ double sum3[MAX_EQN-3];
        
        int j = tid + 3; /* 3 <= j < MAX_EQN */
        /*for (int j = 3; j < MAX_EQN; j++)*/ {
            double UU = E->temp[C[j]];
            sum1[tid] = B1[j]*UU;
            sum2[tid] = B2[j]*UU;
            sum3[tid] = B3[j]*UU;
        }
        __syncthreads();
        
        /* Reduce. */
        if (tid >= 32) {
            sum1[tid-32] += sum1[tid];
            sum2[tid-32] += sum2[tid];
            sum3[tid-32] += sum3[tid];
        }
        __syncthreads();
        for (unsigned int s = 16; s > 0; s >>= 1) {
            if (tid < s) {
                sum1[tid] += sum1[tid + s];
                sum2[tid] += sum2[tid + s];
                sum3[tid] += sum3[tid + s];
            }
            /* XXX: not always necessary */
            __syncthreads();
        }
        
        if (tid == 0) {
            Ad_eqn1 += sum1[0];
            Ad_eqn2 += sum2[0];
            Ad_eqn3 += sum3[0];
            Ad[eqn1] = Ad_eqn1;
            Ad[eqn2] = Ad_eqn2;
            Ad[eqn3] = Ad_eqn3;
            if (!(E->NODE[i] & OFFSIDE)) {
                E->temp[eqn1] = (F[eqn1] - Ad_eqn1)*E->BI[eqn1];
                E->temp[eqn2] = (F[eqn2] - Ad_eqn2)*E->BI[eqn2];
                E->temp[eqn3] = (F[eqn3] - Ad_eqn3)*E->BI[eqn3];
            }
        }
        __syncthreads();
        
    }
    
}


__global__ void gauss_seidel_3(
    struct Some_variables *E,
    double *d0,
    double *Ad
    )
{
    int n = blockIdx.x + 1; /* 1 <= n <= E->lmesh.NNO */
    int doff = blockIdx.y + 1; /* 1 <= doff <= NSD */ 
    unsigned int tid = threadIdx.x; /* 0 <= tid < MAX_EQN */
    
    /* Each block writes one element of Ad and d0 in global memory:
       Ad[eqn], d0[eqn]. */
    int eqn = E->ID[n].doff[doff]; /* XXX: Compute this value? */
    
    __shared__ double sum[MAX_EQN];
    
    int2 *term = E->term + eqn*MAX_EQN;
    int2 pair = term[tid];
    int e = pair.x; /* 1 <= e <= E->lmesh.NNO */
    int i = pair.y; /* 0 <= i < MAX_EQN */
        
    if (i != -1) {
        /* XXX: Compute these values? */
        int eqn1 = E->ID[e].doff[1];
        int eqn2 = E->ID[e].doff[2];
        int eqn3 = E->ID[e].doff[3];
            
        higher_precision *B1, *B2, *B3;
        B1 = E->Eqn_k[1]+(e-1)*MAX_EQN;
        B2 = E->Eqn_k[2]+(e-1)*MAX_EQN;
        B3 = E->Eqn_k[3]+(e-1)*MAX_EQN;
        
        sum[tid] = B1[i]*E->temp[eqn1] +
                   B2[i]*E->temp[eqn2] +
                   B3[i]*E->temp[eqn3];
    } else {
        /* XXX: A considerable number of threads idle here. */
        sum[tid] = 0.0;
    }
    __syncthreads();
    
    /* Reduce the partial sums for this block.
       Based on reduce2() in the CUDA SDK. */
    if (tid >= 32) {
        sum[tid-32] += sum[tid];
    }
    __syncthreads();
    for (unsigned int s = 16; s > 0; s >>= 1) {
        if (tid < s) {
            sum[tid] += sum[tid + s];
        }
        /* XXX: not always necessary */
        __syncthreads();
    }
    
    if (tid == 0) {
        /* Each block writes one element of Ad... */
        Ad[eqn] += sum[0];
        /* ..and one element of d0. */
        d0[eqn] += E->temp[eqn];
    }
}


void host_n_assemble_del2_u(
    struct Some_variables *E,
    double *u, double *Au,
    int strip_bcs
    );

void host_gauss_seidel_0(
    struct Some_variables *E,
    double *d0,
    double *Ad
    );

void host_gauss_seidel_1(
    struct Some_variables *E,
    struct Some_variables *s_E,
    double *F, double *Ad
    );

void host_gauss_seidel_2(
    struct Some_variables *E,
    struct Some_variables *s_E,
    double *F, double *Ad
    );

void host_gauss_seidel_3(
    struct Some_variables *E,
    struct Some_variables *s_E,
    double *d0,
    double *Ad
    );


void do_gauss_seidel(
    struct Some_variables *E,
    double *d0,
    double *F, double *Ad,
    double acc,
    int *cycles,
    int guess
    )
{

    int count, steps;

    steps=*cycles;

    /* pointers to device memory */
    struct Some_variables *d_E = 0;
    double *d_d0 = 0, *d_F = 0, *d_Ad = 0;
    
    /* construct 'E' on the device */
    struct Some_variables s_E;
    construct_E(&d_E, &s_E, E);
    
    int neq = E->lmesh.NEQ;
    
    /* allocate memory on the device */
    hipMalloc((void**)&d_d0, (1+neq)*sizeof(double));
    hipMalloc((void**)&d_F, neq*sizeof(double));
    hipMalloc((void**)&d_Ad, (1+neq)*sizeof(double));
    
    /* copy input to the device */
    hipMemcpy(d_F, F, neq*sizeof(double), hipMemcpyHostToDevice);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    static float totalElapsedTime = 0.0;
    
    if (guess) {
        /* copy more input to the device */
        d0[E->lmesh.NEQ] = 0.0; /* normally done by n_assemble_del2_u() */
        hipMemcpy(d_d0, d0, (1+neq)*sizeof(double), hipMemcpyHostToDevice);
        
        dim3 block(MAX_EQN, 1, 1);
        dim3 grid(E->lmesh.NNO, NSD, 1);
        if (1) n_assemble_del2_u<<< grid, block >>>(d_E, d_d0, d_Ad, 1);
        else host_n_assemble_del2_u(E, d_d0, d_Ad, 1);
    
    } else {
        const int blockSize = 64;
        const int dim = E->lmesh.NEQ;
        int gridSize = dim / blockSize;
        if (dim % blockSize)
            ++gridSize;
        
        dim3 block(blockSize, 1, 1);
        dim3 grid(gridSize, 1, 1);
        gauss_seidel_0<<< grid, block >>>(E->lmesh.NEQ, d_d0, d_Ad);
    }
    
    for (count = 0; count < steps; ++count) {
        {
            const int blockSize = 64;
            const int dim = E->lmesh.NNO;
            int gridSize = dim / blockSize;
            if (dim % blockSize)
                ++gridSize;
            
            dim3 block(blockSize, 1, 1);
            dim3 grid(gridSize, NSD, 1);
            
            hipEventRecord(start, 0);
            
            gauss_seidel_1<<< grid, block >>>(d_E, E->lmesh.NEQ, E->lmesh.NNO, d_F, d_Ad);
            
            hipEventRecord(stop, 0);
            if (0) {
                hipEventSynchronize(stop);
                float elapsedTime;
                hipEventElapsedTime(&elapsedTime, start, stop);
                totalElapsedTime += elapsedTime;
                if (count == steps - 1) {
                    printf("gauss_seidel_0 %d\t%f\t%f\n", count, elapsedTime, totalElapsedTime);
                }
            }
        }
        
        if (1) {
            dim3 block(MAX_EQN - 3, NSD, 1);
            dim3 grid(1, 1, 1);
            gauss_seidel_2a<<< grid, block >>>(d_E, d_F, d_Ad);
        }
        if (0) {
            dim3 block(MAX_EQN - 3, 1, 1);
            dim3 grid(1, 1, 1);
            gauss_seidel_2b<<< grid, block >>>(d_E, d_F, d_Ad);
        }
        if (0) host_gauss_seidel_2(E, &s_E, d_F, d_Ad);
        
        /* Ad on boundaries differs after the following operation */
        {
            dim3 block(MAX_EQN, 1, 1);
            dim3 grid(E->lmesh.NNO, NSD, 1);
            if (1) gauss_seidel_3<<< grid, block >>>(d_E, d_d0, d_Ad);
            else host_gauss_seidel_3(E, &s_E, d_d0, d_Ad);            
        }
    }
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    /* wait for completion */
    if (hipDeviceSynchronize() != hipSuccess) {
        assert(0 && "something went wrong");
    }
    
    /* copy output from device */
    hipMemcpy(Ad, d_Ad, (1+neq)*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(d0, d_d0, (1+neq)*sizeof(double), hipMemcpyDeviceToHost);
    
    /* free device memory */
    hipFree(d_d0);
    hipFree(d_F);
    hipFree(d_Ad);
    
    destroy_E(d_E, &s_E);
    
    *cycles=count;
    
    return;
}


/*------------------------------------------------------------------------*/

static void assert_assumptions(struct All_variables *E, int level) {
    
    assert(E->control.NMULTIGRID);
    
    assert(E->sphere.caps_per_proc == CAPS_PER_PROC);
    
    assert(E->mesh.nsd == NSD);
    
    assert(E->parallel.nproc == 1);
}

static int2 *collect_terms(
    struct Some_variables *E
    )
{
    /* Map out how to parallelize "Au[C[i]] += ..." and "Ad[C[j]] += ...". */
    
    const int neq = E->lmesh.NEQ;
    const int nno = E->lmesh.NNO;
    
    int2 *term = (int2 *)malloc((neq+1) * MAX_EQN * sizeof(int2));
    
    for (int e = 0; e <= neq; e++) {
        int2 *term = E->term + e*MAX_EQN;
        for (int j = 0; j < MAX_EQN; j++) {
            term[j].x = -1;
            term[j].y = -1;
        }
    }
    
    for (int e = 1; e <= nno; e++) {
        int *C = E->Node_map + (e-1)*MAX_EQN;
        for (int i = 0; i < MAX_EQN; i++) {
            int2 *term = E->term + C[i]*MAX_EQN;
            int j;
            for (j = 0; j < MAX_EQN; j++) {
                if (term[j].x == -1) {
                    term[j].x = e;
                    term[j].y = i;
                    break;
                }
            }
            assert(C[i] == neq || j < MAX_EQN);
        }
    }
    
    return term;
}


extern "C" void gauss_seidel(
    struct All_variables *E,
    double **d0,
    double **F, double **Ad,
    double acc,
    int *cycles,
    int level,
    int guess
    )
{
    struct Some_variables kE;
    
    assert_assumptions(E, level);
    
    /* initialize 'Some_variables' with 'All_variables' */
    
    kE.num_zero_resid = E->num_zero_resid[level][M];
    kE.zero_resid = E->zero_resid[level][M];
    
    kE.lmesh.NEQ = E->lmesh.NEQ[level];
    kE.lmesh.NNO = E->lmesh.NNO[level];
    
    kE.ID    = E->ID[level][M];
    
    kE.Eqn_k[0] = 0;
    kE.Eqn_k[1] = E->Eqn_k1[level][M];
    kE.Eqn_k[2] = E->Eqn_k2[level][M];
    kE.Eqn_k[3] = E->Eqn_k3[level][M];
    kE.Node_map = E->Node_map[level][M];
    
    kE.BI = E->BI[level][M];
    
    kE.temp = E->temp[M];
    
    kE.NODE = E->NODE[level][M];
    
    static int2 *term[MAX_LEVELS];
    if (!term[level])
        term[level] = collect_terms(&kE);
    kE.term = term[level];
    
    do_gauss_seidel(
        &kE,
        d0[M],
        F[M], Ad[M],
        acc,
        cycles,
        guess
        );
}



/*------------------------------------------------------------------------*/
/*------------------------------------------------------------------------*/
/*------------------------------------------------------------------------*/

void host_strip_bcs_from_residual(
    struct Some_variables *E,
    double *Res
    )
{
    for(int i = 1; i <= E->num_zero_resid; i++)
        Res[E->zero_resid[i]] = 0.0;
}


void host_n_assemble_del2_u(
    struct Some_variables *E,
    double *d_u, double *d_Au,
    int strip_bcs
    )
{
    hipDeviceSynchronize();
    
    const int neq = E->lmesh.NEQ;
    const int nno = E->lmesh.NNO;
    
    double *u = (double *)malloc((1+neq)*sizeof(double));
    hipMemcpy(u, d_u, (1+neq)*sizeof(double), hipMemcpyDeviceToHost);
    
    double *Au = (double *)malloc((1+neq)*sizeof(double));
    
    for (int e = 0; e <= neq; e++) {
        Au[e] = 0.0;
    }
    
    u[neq] = 0.0;
    
    for (int e = 1; e <= nno; e++) {
        
        int eqn1 = E->ID[e].doff[1];
        int eqn2 = E->ID[e].doff[2];
        int eqn3 = E->ID[e].doff[3];
        
        double U1 = u[eqn1];
        double U2 = u[eqn2];
        double U3 = u[eqn3];
        
        int *C = E->Node_map + (e-1)*MAX_EQN;
        
        higher_precision *B1,*B2,*B3;
        B1 = E->Eqn_k[1] + (e-1)*MAX_EQN;
        B2 = E->Eqn_k[2] + (e-1)*MAX_EQN;
        B3 = E->Eqn_k[3] + (e-1)*MAX_EQN;
        
        for (int i = 3; i < MAX_EQN; i++)  {
            double UU = u[C[i]];
            Au[eqn1] += B1[i]*UU;
            Au[eqn2] += B2[i]*UU;
            Au[eqn3] += B3[i]*UU;
        }
        for (int i = 0; i < MAX_EQN; i++) {
            Au[C[i]] += B1[i]*U1 +
                        B2[i]*U2 +
                        B3[i]*U3;
        }
    }
    
    if (strip_bcs)
        host_strip_bcs_from_residual(E,Au);
    
    hipMemcpy(d_Au, Au, (1+neq)*sizeof(double), hipMemcpyHostToDevice);
    
    free(u);
    free(Au);
    
    return;
}


void host_gauss_seidel_0(
    struct Some_variables *E,
    double *d_d0,
    double *d_Ad
    )
{
    hipDeviceSynchronize();
    
    const int neq = E->lmesh.NEQ;
    
    double *d0 = (double *)malloc((1+neq)*sizeof(double));
    double *Ad = (double *)malloc((1+neq)*sizeof(double));
    
    const double zeroo = 0.0;
    
    for (int i = 0; i < neq; i++) {
        d0[i] = Ad[i] = zeroo;
    }
    
    hipMemcpy(d_d0, d0, (1+neq)*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_Ad, Ad, (1+neq)*sizeof(double), hipMemcpyHostToDevice);
    
    free(d0);
    free(Ad);
}


void host_gauss_seidel_1(
    struct Some_variables *E,
    struct Some_variables *s_E,
    double *d_F, double *d_Ad
    )
{
    hipDeviceSynchronize();
    
    const int neq = E->lmesh.NEQ;
    const int nno = E->lmesh.NNO;
    
    double *F = (double *)malloc(neq*sizeof(double));
    hipMemcpy(F, d_F, neq*sizeof(double), hipMemcpyDeviceToHost);
    
    double *Ad = (double *)malloc((1+neq)*sizeof(double));
    hipMemcpy(Ad, d_Ad, (1+neq)*sizeof(double), hipMemcpyDeviceToHost);
    
    const double zeroo = 0.0;
    
    for (int j = 0; j <= neq; j++) {
        E->temp[j] = zeroo;
    }
    
    Ad[neq] = zeroo;
    
    for (int i = 1; i <= nno; i++) {
        if (E->NODE[i] & OFFSIDE) {
            int eqn1 = E->ID[i].doff[1];
            int eqn2 = E->ID[i].doff[2];
            int eqn3 = E->ID[i].doff[3];
            E->temp[eqn1] = (F[eqn1] - Ad[eqn1])*E->BI[eqn1];
            E->temp[eqn2] = (F[eqn2] - Ad[eqn2])*E->BI[eqn2];
            E->temp[eqn3] = (F[eqn3] - Ad[eqn3])*E->BI[eqn3];
        }
    }
    
    /* Ad[neq] */
    hipMemcpy(d_Ad + neq, Ad + neq, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(s_E->temp, E->temp, (neq+1)*sizeof(double), hipMemcpyHostToDevice);
    
    free(F);
    free(Ad);
}


void host_gauss_seidel_2(
    struct Some_variables *E,
    struct Some_variables *s_E,
    double *d_F, double *d_Ad
    )
{
    hipDeviceSynchronize();
    
    const int neq = E->lmesh.NEQ;
    const int nno = E->lmesh.NNO;
    
    double *F = (double *)malloc(neq*sizeof(double));
    hipMemcpy(F, d_F, neq*sizeof(double), hipMemcpyDeviceToHost);
    
    double *Ad = (double *)malloc((1+neq)*sizeof(double));
    hipMemcpy(Ad, d_Ad, (1+neq)*sizeof(double), hipMemcpyDeviceToHost);
    
    hipMemcpy(E->temp, s_E->temp, (neq+1)*sizeof(double), hipMemcpyDeviceToHost);
    
    for (int i = 1; i <= nno; i++) {
            
        int eqn1 = E->ID[i].doff[1];
        int eqn2 = E->ID[i].doff[2];
        int eqn3 = E->ID[i].doff[3];
            
        int *C = E->Node_map + (i-1)*MAX_EQN;
            
        higher_precision *B1,*B2,*B3;
        B1 = E->Eqn_k[1] + (i-1)*MAX_EQN;
        B2 = E->Eqn_k[2] + (i-1)*MAX_EQN;
        B3 = E->Eqn_k[3] + (i-1)*MAX_EQN;
            
        /* Ad on boundaries differs after the following operation, but
           no communications are needed yet, because boundary Ad will
           not be used for the G-S iterations for interior nodes */
            
        for (int j = 3; j < MAX_EQN; j++) {
            double UU = E->temp[C[j]];
            Ad[eqn1] += B1[j]*UU;
            Ad[eqn2] += B2[j]*UU;
            Ad[eqn3] += B3[j]*UU;
        }
            
        if (!(E->NODE[i] & OFFSIDE)) {
            E->temp[eqn1] = (F[eqn1] - Ad[eqn1])*E->BI[eqn1];
            E->temp[eqn2] = (F[eqn2] - Ad[eqn2])*E->BI[eqn2];
            E->temp[eqn3] = (F[eqn3] - Ad[eqn3])*E->BI[eqn3];
        }
            
    }
            
    hipMemcpy(d_Ad, Ad, (1+neq)*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(s_E->temp, E->temp, (neq+1)*sizeof(double), hipMemcpyHostToDevice);
    
    free(F);
    free(Ad);
}


void host_gauss_seidel_3(
    struct Some_variables *E,
    struct Some_variables *s_E,
    double *d_d0,
    double *d_Ad
    )
{
    hipDeviceSynchronize();
    
    const int neq = E->lmesh.NEQ;
    const int nno = E->lmesh.NNO;
    
    double *d0 = (double *)malloc((1+neq)*sizeof(double));
    double *Ad = (double *)malloc((1+neq)*sizeof(double));
    
    hipMemcpy(d0, d_d0, (1+neq)*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(Ad, d_Ad, (1+neq)*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(E->temp, s_E->temp, (neq+1)*sizeof(double), hipMemcpyDeviceToHost);
    
    for (int i = 1; i <= nno; i++) {
            
        int eqn1 = E->ID[i].doff[1];
        int eqn2 = E->ID[i].doff[2];
        int eqn3 = E->ID[i].doff[3];
            
        int *C = E->Node_map + (i-1)*MAX_EQN;
            
        higher_precision *B1,*B2,*B3;
        B1 = E->Eqn_k[1] + (i-1)*MAX_EQN;
        B2 = E->Eqn_k[2] + (i-1)*MAX_EQN;
        B3 = E->Eqn_k[3] + (i-1)*MAX_EQN;
            
        /* Ad on boundaries differs after the following operation */
        for (int j = 0; j < MAX_EQN; j++) {
            Ad[C[j]] += B1[j]*E->temp[eqn1] +
                        B2[j]*E->temp[eqn2] +
                        B3[j]*E->temp[eqn3];
        }
            
        d0[eqn1] += E->temp[eqn1];
        d0[eqn2] += E->temp[eqn2];
        d0[eqn3] += E->temp[eqn3];
            
    }
    
    hipMemcpy(d_d0, d0, (1+neq)*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_Ad, Ad, (1+neq)*sizeof(double), hipMemcpyHostToDevice);
    
    free(d0);
    free(Ad);
}
